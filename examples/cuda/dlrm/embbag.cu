#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <algorithm>
#include <fstream>
#include <string>
#include <sstream>
#define FEATSIZE 256
#define FEATSIZELOG2 8
#define MAX_BATCHES 256

__global__ void EmbeddingBag(
    const int *input, 
    const int *offsets,
    const float *weight,
    float *output,  
    int64_t numBags
    ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t bag = (int64_t)idx >> FEATSIZELOG2;  
    int64_t featureDim = (FEATSIZE - 1) & (int64_t)idx;
    
    int64_t begin = offsets[bag];
    int64_t end = offsets[bag+1];
    const float *weightFeat = weight + featureDim;
    float weightFeatSum = 0;
    
    for (int64_t emb = begin; emb < end; emb++) {
        const int64_t weightRow = input[emb];
        float weightValue = weightFeat[weightRow * FEATSIZE];
        weightFeatSum += weightValue;
    }
    output[bag * FEATSIZE + featureDim] = weightFeatSum;
}

int main() {
    const int numBags = 32;  // batchsize 1 4 8 10 20 30 32 256
    const int numLookup = 80;
    const int value_size = numLookup * numBags;
    const int featureSize = FEATSIZE;
    const int numRows = 1000000;

    int input[MAX_BATCHES * numLookup];
    int offsets[numBags + 1];
    
    std::ifstream file("../../data/kaggle_emb/emb_0.txt");
    if (!file.is_open()) {
        std::cerr << "Cannot open the file" << std::endl;
        return 1;
    }

    int number;
    int idx = 0;
    std::string line;
    for (int b_idx=0; b_idx < MAX_BATCHES; b_idx++){
        std::getline(file, line);
        std::stringstream ss(line);
        for (int f_idx = 0; f_idx < numLookup; f_idx++) {
            ss >> number;
            input[idx++] = number;
        }
    }

    for (int i=0; i<numBags + 1; i++)
        offsets[i] = numLookup * i;

    int *d_input, *d_offsets;
    float *d_weight, *d_output;
    hipMalloc(&d_input, value_size * sizeof(int));
    hipMalloc(&d_offsets, (numBags + 1) * sizeof(int));
    hipMalloc(&d_weight, numRows * featureSize * sizeof(float));
    hipMalloc(&d_output, numBags * featureSize * sizeof(float));

    
    hipMemcpy(d_offsets, offsets, (numBags + 1) * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = numBags * featureSize / blockSize;
    printf("GRID = %d, BLOCK = %d\n", gridSize, blockSize);
    for(int i = 0; i < MAX_BATCHES / numBags; i++) {
        hipMemcpy(d_input, input + i * value_size, value_size * sizeof(int), hipMemcpyHostToDevice);
        EmbeddingBag<<<gridSize, blockSize>>>(
        d_input, d_offsets, d_weight, d_output, numBags);
    }
    

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    hipFree(d_input);
    hipFree(d_offsets);
    hipFree(d_weight);
    hipFree(d_output);

    return 0;
}