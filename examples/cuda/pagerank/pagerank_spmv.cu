/************************************************************************************\ 
 *                                                                                  *
 * Copyright � 2014 Advanced Micro Devices, Inc.                                    *
 * Copyright (c) 2015 Mark D. Hill and David A. Wood                                *
 * All rights reserved.                                                             *
 *                                                                                  *
 * Redistribution and use in source and binary forms, with or without               *
 * modification, are permitted provided that the following are met:                 *
 *                                                                                  *
 * You must reproduce the above copyright notice.                                   *
 *                                                                                  *
 * Neither the name of the copyright holder nor the names of its contributors       *
 * may be used to endorse or promote products derived from this software            *
 * without specific, prior, written permission from at least the copyright holder.  *
 *                                                                                  *
 * You must include the following terms in your license and/or other materials      *
 * provided with the software.                                                      *
 *                                                                                  *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"      *
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE        *
 * IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT, AND FITNESS FOR A       *
 * PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER        *
 * OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,         *
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT  *
 * OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS      *
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN          *
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
 * IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY   *
 * OF SUCH DAMAGE.                                                                  *
 *                                                                                  *
 * Without limiting the foregoing, the software may implement third party           *
 * technologies for which you must obtain licenses from parties other than AMD.     *
 * You agree that AMD has not obtained or conveyed to you, and that you shall       *
 * be responsible for obtaining the rights to use and/or distribute the applicable  *
 * underlying intellectual property rights related to the third party technologies. *
 * These third party technologies are not licensed hereunder.                       *
 *                                                                                  *
 * If you use the software (in whole or in part), you shall adhere to all           *
 * applicable U.S., European, and other export laws, including but not limited to   *
 * the U.S. Export Administration Regulations ("EAR"�) (15 C.F.R Sections 730-774),  *
 * and E.U. Council Regulation (EC) No 428/2009 of 5 May 2009.  Further, pursuant   *
 * to Section 740.6 of the EAR, you hereby certify that, except pursuant to a       *
 * license granted by the United States Department of Commerce Bureau of Industry   *
 * and Security or as otherwise permitted pursuant to a License Exception under     *
 * the U.S. Export Administration Regulations ("EAR"), you will not (1) export,     *
 * re-export or release to a national of a country in Country Groups D:1, E:1 or    *
 * E:2 any restricted technology, software, or source code you receive hereunder,   *
 * or (2) export to Country Groups D:1, E:1 or E:2 the direct product of such       *
 * technology or software, if such foreign produced direct product is subject to    *
 * national security controls as identified on the Commerce Control List (currently *
 * found in Supplement 1 to Part 774 of EAR).  For the most current Country Group   *
 * listings, or for additional information about the EAR or your obligations under  *
 * those regulations, please refer to the U.S. Bureau of Industry and Security's    *
 * website at http://www.bis.doc.gov/.                                              *
 *                                                                                  *
\************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "./graph_parser/parse.h"
#include "./graph_parser/util.h"
#include "kernel_spmv.cu"
#include <hip/hip_runtime_api.h>
#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

// Iteration count
#define ITER 1

void print_vectorf(float *vector, int num);

int main(int argc, char **argv)
{
    char *tmpchar;

    int num_nodes;
    int num_edges;
    int file_format = 1;
    bool directed = 0;
    char file_name[100] = "\0";

    hipError_t err = hipSuccess;

    if (argc == 3) {
        tmpchar = argv[1]; // Graph inputfile
        file_format = atoi(argv[2]);
    } else {
        fprintf(stderr, "You did something wrong!\n");
        exit(1);
    }

    // Allocate the csr structure
    csr_array *csr;

    // Parse graph files into csr structure
    if (file_format == 1) {
       csr = parseMetis_transpose(tmpchar, &num_nodes, &num_edges, directed);
    } else if (file_format == 0) {
       csr = parseCOO_transpose(tmpchar, &num_nodes, &num_edges, directed);
    } else {
       printf("reserve for future");
       exit(1);
    }

    FILE *kernel1_fp = fopen("kernel1_input.txt", "w");
    FILE *kernel2_fp0 = fopen("kernel2_input0.txt", "w");
    FILE *kernel2_fp1 = fopen("kernel2_input1.txt", "w");
    FILE *kernel2_fp2 = fopen("kernel2_input2.txt", "w");
    FILE *kernel2_fp3 = fopen("kernel2_input3.txt", "w");
    FILE *kernel2_fp4 = fopen("kernel2_input4.txt", "w");
    FILE *kernel2_fp5 = fopen("kernel2_input5.txt", "w");
    FILE *kernel2_fp6 = fopen("kernel2_input6.txt", "w");
    FILE *kernel2_fp7 = fopen("kernel2_input7.txt", "w");
    FILE *kernel2_fp8 = fopen("kernel2_input8.txt", "w");
    FILE *kernel2_fp9 = fopen("kernel2_input9.txt", "w");
    FILE *kernel2_fp10 = fopen("kernel2_input10.txt", "w");
    FILE *kernel2_fp11 = fopen("kernel2_input11.txt", "w");
    FILE *kernel2_fp12 = fopen("kernel2_input12.txt", "w");
    FILE *kernel2_fp13 = fopen("kernel2_input13.txt", "w");
    FILE *kernel2_fp14 = fopen("kernel2_input14.txt", "w");
    FILE *kernel2_fp15 = fopen("kernel2_input15.txt", "w");
    FILE *kernel2_fp16 = fopen("kernel2_input16.txt", "w");
    FILE *kernel2_fp17 = fopen("kernel2_input17.txt", "w");
    FILE *kernel2_fp18 = fopen("kernel2_input18.txt", "w");
    FILE *kernel2_fp19 = fopen("kernel2_input19.txt", "w");
    
    FILE *kernel3_fp0 = fopen("kernel3_input0.txt", "w");
    FILE *kernel3_fp1 = fopen("kernel3_input1.txt", "w");
    FILE *kernel3_fp2 = fopen("kernel3_input2.txt", "w");
    FILE *kernel3_fp3 = fopen("kernel3_input3.txt", "w");
    FILE *kernel3_fp4 = fopen("kernel3_input4.txt", "w");
    FILE *kernel3_fp5 = fopen("kernel3_input5.txt", "w");
    FILE *kernel3_fp6 = fopen("kernel3_input6.txt", "w");
    FILE *kernel3_fp7 = fopen("kernel3_input7.txt", "w");
    FILE *kernel3_fp8 = fopen("kernel3_input8.txt", "w");
    FILE *kernel3_fp9 = fopen("kernel3_input9.txt", "w");
    FILE *kernel3_fp10 = fopen("kernel3_input10.txt", "w");
    FILE *kernel3_fp11 = fopen("kernel3_input11.txt", "w");
    FILE *kernel3_fp12 = fopen("kernel3_input12.txt", "w");
    FILE *kernel3_fp13 = fopen("kernel3_input13.txt", "w");
    FILE *kernel3_fp14 = fopen("kernel3_input14.txt", "w");
    FILE *kernel3_fp15 = fopen("kernel3_input15.txt", "w");
    FILE *kernel3_fp16 = fopen("kernel3_input16.txt", "w");
    FILE *kernel3_fp17 = fopen("kernel3_input17.txt", "w");
    FILE *kernel3_fp18 = fopen("kernel3_input18.txt", "w");
    FILE *kernel3_fp19 = fopen("kernel3_input19.txt", "w");

    fprintf(kernel1_fp, "row_array\n");
    fprintf(kernel2_fp0, "row_array\n");
    fprintf(kernel2_fp1, "row_array\n");
    fprintf(kernel2_fp2, "row_array\n");
    fprintf(kernel2_fp3, "row_array\n");
    fprintf(kernel2_fp4, "row_array\n");
    fprintf(kernel2_fp5, "row_array\n");
    fprintf(kernel2_fp6, "row_array\n");
    fprintf(kernel2_fp7, "row_array\n");
    fprintf(kernel2_fp8, "row_array\n");
    fprintf(kernel2_fp9, "row_array\n");
    fprintf(kernel2_fp10, "row_array\n");
    fprintf(kernel2_fp11, "row_array\n");
    fprintf(kernel2_fp12, "row_array\n");
    fprintf(kernel2_fp13, "row_array\n");
    fprintf(kernel2_fp14, "row_array\n");
    fprintf(kernel2_fp15, "row_array\n");
    fprintf(kernel2_fp16, "row_array\n");
    fprintf(kernel2_fp17, "row_array\n");
    fprintf(kernel2_fp18, "row_array\n");
    fprintf(kernel2_fp19, "row_array\n");
    for (int i = 0; i < num_nodes + 1; i++){
        fprintf(kernel1_fp, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp0, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp1, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp2, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp3, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp4, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp5, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp6, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp7, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp8, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp9, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp10, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp11, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp12, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp13, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp14, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp15, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp16, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp17, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp18, "%d\n", csr->row_array[i]);
        fprintf(kernel2_fp19, "%d\n", csr->row_array[i]);
    }

    fprintf(kernel1_fp, "col_array\n");
    fprintf(kernel2_fp0, "col_array\n");
    fprintf(kernel2_fp1, "col_array\n");
    fprintf(kernel2_fp2, "col_array\n");
    fprintf(kernel2_fp3, "col_array\n");
    fprintf(kernel2_fp4, "col_array\n");
    fprintf(kernel2_fp5, "col_array\n");
    fprintf(kernel2_fp6, "col_array\n");
    fprintf(kernel2_fp7, "col_array\n");
    fprintf(kernel2_fp8, "col_array\n");
    fprintf(kernel2_fp9, "col_array\n");
    fprintf(kernel2_fp10, "col_array\n");
    fprintf(kernel2_fp11, "col_array\n");
    fprintf(kernel2_fp12, "col_array\n");
    fprintf(kernel2_fp13, "col_array\n");
    fprintf(kernel2_fp14, "col_array\n");
    fprintf(kernel2_fp15, "col_array\n");
    fprintf(kernel2_fp16, "col_array\n");
    fprintf(kernel2_fp17, "col_array\n");
    fprintf(kernel2_fp18, "col_array\n");
    fprintf(kernel2_fp19, "col_array\n");
    for (int i = 0; i < num_edges; i++){
        fprintf(kernel1_fp, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp0, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp1, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp2, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp3, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp4, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp5, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp6, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp7, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp8, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp9, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp10, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp11, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp12, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp13, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp14, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp15, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp16, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp17, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp18, "%d\n", csr->col_array[i]);
        fprintf(kernel2_fp19, "%d\n", csr->col_array[i]);
    }
    fprintf(kernel1_fp, "col_cnt\n");
    for (int i = 0; i < num_nodes; i++)
        fprintf(kernel1_fp, "%d\n", csr->col_cnt[i]);
    fclose(kernel1_fp);
    // Allocate rank_arrays
    float *pagerank_array = (float *)malloc(num_nodes * sizeof(float));
    if (!pagerank_array) fprintf(stderr, "malloc failed page_rank_array\n");
    float *pagerank_array2 = (float *)malloc(num_nodes * sizeof(float));
    if (!pagerank_array2) fprintf(stderr, "malloc failed page_rank_array2\n");
    float *data = (float *)malloc(num_edges * sizeof(float));
    // if (!pagerank_array2) fprintf(stderr, "malloc failed page_rank_array2\n");
    int *row_d;
    int *col_d;
    float *data_d;

    float *pagerank1_d;
    float *pagerank2_d;
    int *col_cnt_d;

    // Create device-side buffers for the graph
    err = hipMalloc(&row_d, (num_nodes + 1) * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc row_d (size:%d) => %s\n",  num_nodes, hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&col_d, num_edges * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc col_d (size:%d) => %s\n",  num_edges, hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&data_d, num_edges * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc data_d (size:%d) => %s\n", num_edges, hipGetErrorString(err));
        return -1;
    }

    // Create buffers for pagerank
    err = hipMalloc(&pagerank1_d, num_nodes * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc pagerank1_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&pagerank2_d, num_nodes * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc pagerank2_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&col_cnt_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc col_cnt_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    double timer1 = gettime();

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

    // Copy the data to the device-side buffers
    err = hipMemcpy(row_d, csr->row_array, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR:#endif hipMemcpy row_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(col_d, csr->col_array, num_edges * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy col_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(col_cnt_d, csr->col_cnt, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy col_cnt_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    // Set up work dimensions
    int block_size = 64;
    // int num_blocks = (num_nodes + block_size - 1) / block_size;
    int rows_per_block = block_size / 32;
    int num_blocks = (num_nodes + rows_per_block - 1) / rows_per_block;

    dim3 threads(block_size, 1, 1);
    dim3 grid(num_blocks, 1, 1);

    double timer3 = gettime();
    hipProfilerStart();
    // Launch the initialization kernel
    inibuffer <<<grid, threads>>>(pagerank1_d, pagerank2_d, num_nodes);
    err = hipMemcpy(pagerank_array, pagerank1_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    err = hipMemcpy(pagerank_array2, pagerank2_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    FILE *kernel0_fp = fopen("kernel0_output.txt", "w");
    fprintf(kernel0_fp, "pagerank1\n");
    for (int i = 0; i < num_nodes; i++)
        fprintf(kernel0_fp, "%.16f\n", pagerank_array[i]);
    fprintf(kernel0_fp, "pagerank2\n");
    for (int i = 0; i < num_nodes; i++)
        fprintf(kernel0_fp, "%.16f\n", pagerank_array2[i]);
    fclose(kernel0_fp);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipLaunchByPtr failed (%s)\n", hipGetErrorString(err));
        return -1;
    }

    // Initialize the CSR
    inicsr <<<grid, threads>>>(row_d, col_d, data_d, col_cnt_d, num_nodes,
                               num_edges);
    hipDeviceSynchronize();
    
    err = hipMemcpy(data, data_d, num_edges * sizeof(float), hipMemcpyDeviceToHost);
    FILE *kernel1_out = fopen("kernel1_output.txt", "w");
    fprintf(kernel2_fp0, "data_array\n");
    fprintf(kernel2_fp1, "data_array\n");
    fprintf(kernel2_fp2, "data_array\n");
    fprintf(kernel2_fp3, "data_array\n");
    fprintf(kernel2_fp4, "data_array\n");
    fprintf(kernel2_fp5, "data_array\n");
    fprintf(kernel2_fp6, "data_array\n");
    fprintf(kernel2_fp7, "data_array\n");
    fprintf(kernel2_fp8, "data_array\n");
    fprintf(kernel2_fp9, "data_array\n");
    fprintf(kernel2_fp10, "data_array\n");
    fprintf(kernel2_fp11, "data_array\n");
    fprintf(kernel2_fp12, "data_array\n");
    fprintf(kernel2_fp13, "data_array\n");
    fprintf(kernel2_fp14, "data_array\n");
    fprintf(kernel2_fp15, "data_array\n");
    fprintf(kernel2_fp16, "data_array\n");
    fprintf(kernel2_fp17, "data_array\n");
    fprintf(kernel2_fp18, "data_array\n");
    fprintf(kernel2_fp19, "data_array\n");

    for (int i = 0; i < num_edges; i++){
       fprintf(kernel1_out, "%.16f\n", data[i]);
       fprintf(kernel2_fp0, "%.16f\n", data[i]);
       fprintf(kernel2_fp1, "%.16f\n", data[i]);
       fprintf(kernel2_fp2, "%.16f\n", data[i]);
       fprintf(kernel2_fp3, "%.16f\n", data[i]);
       fprintf(kernel2_fp4, "%.16f\n", data[i]);
       fprintf(kernel2_fp5, "%.16f\n", data[i]);
       fprintf(kernel2_fp6, "%.16f\n", data[i]);
       fprintf(kernel2_fp7, "%.16f\n", data[i]);
       fprintf(kernel2_fp8, "%.16f\n", data[i]);
       fprintf(kernel2_fp9, "%.16f\n", data[i]);
       fprintf(kernel2_fp10, "%.16f\n", data[i]);
       fprintf(kernel2_fp11, "%.16f\n", data[i]);
       fprintf(kernel2_fp12, "%.16f\n", data[i]);
       fprintf(kernel2_fp13, "%.16f\n", data[i]);
       fprintf(kernel2_fp14, "%.16f\n", data[i]);
       fprintf(kernel2_fp15, "%.16f\n", data[i]);
       fprintf(kernel2_fp16, "%.16f\n", data[i]);
       fprintf(kernel2_fp17, "%.16f\n", data[i]);
       fprintf(kernel2_fp18, "%.16f\n", data[i]);
       fprintf(kernel2_fp19, "%.16f\n", data[i]);
    }
    fclose(kernel1_out);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipLaunchByPtr failed (%s)\n", hipGetErrorString(err));
        return -1;
    }

    // Run PageRank for some iter. TO: convergence determination
    for (int i = 0; i < ITER; i++) {
        err = hipMemcpy(pagerank_array , pagerank1_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        err = hipMemcpy(pagerank_array2, pagerank2_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        if(i == 0){
            fprintf(kernel2_fp0, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp0, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp0, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp0, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 1){
            fprintf(kernel2_fp1, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp1, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp1, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp1, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 2){
            fprintf(kernel2_fp2, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp2, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp2, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp2, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 3){
            fprintf(kernel2_fp3, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp3, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp3, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp3, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 4){
            fprintf(kernel2_fp4, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp4, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp4, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp4, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 5){
            fprintf(kernel2_fp5, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp5, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp5, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp5, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 6){
            fprintf(kernel2_fp6, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp6, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp6, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp6, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 7){
            fprintf(kernel2_fp7, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp7, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp7, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp7, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 8){
            fprintf(kernel2_fp8, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp8, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp8, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp8, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 9){
            fprintf(kernel2_fp9, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp9, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp9, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp9, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 10){
            fprintf(kernel2_fp10, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp10, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp10, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp10, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 11){
            fprintf(kernel2_fp11, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp11, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp11, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp11, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 12){
            fprintf(kernel2_fp12, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp12, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp12, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp12, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 13){
            fprintf(kernel2_fp13, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp13, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp13, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp13, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 14){
            fprintf(kernel2_fp14, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp14, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp14, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp14, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 15){
            fprintf(kernel2_fp15, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp15, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp15, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp15, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 16){
            fprintf(kernel2_fp16, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp16, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp16, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp16, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 17){
            fprintf(kernel2_fp17, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp17, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp17, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp17, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 18){
            fprintf(kernel2_fp18, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp18, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp18, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp18, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 19){
            fprintf(kernel2_fp19, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp19, "%.16f\n", pagerank_array[i]);
            fprintf(kernel2_fp19, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel2_fp19, "%.16f\n", pagerank_array2[i]);
        }
        // Launch pagerank kernel 1
        spmv_csr_scalar_kernel <<<grid, threads>>>(num_nodes, row_d, col_d,
                                                   data_d, pagerank1_d,
                                                   pagerank2_d);
        sprintf(file_name, "kernel2_output%d.txt", i);
        err = hipMemcpy(pagerank_array , pagerank1_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        err = hipMemcpy(pagerank_array2, pagerank2_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        FILE* kernel2_out = fopen(file_name, "w");
        for (int j = 0; j < num_nodes; j++)
            fprintf(kernel2_out, "%.16f\n", pagerank_array2[j]);
        fclose(kernel2_out);

        /* kernel 3*/
        if(i == 0){
            fprintf(kernel3_fp0, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp0, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp0, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp0, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 1){
            fprintf(kernel3_fp1, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp1, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp1, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp1, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 2){
            fprintf(kernel3_fp2, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp2, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp2, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp2, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 3){
            fprintf(kernel3_fp3, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp3, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp3, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp3, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 4){
            fprintf(kernel3_fp4, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp4, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp4, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp4, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 5){
            fprintf(kernel3_fp5, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp5, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp5, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp5, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 6){
            fprintf(kernel3_fp6, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp6, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp6, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp6, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 7){
            fprintf(kernel3_fp7, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp7, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp7, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp7, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 8){
            fprintf(kernel3_fp8, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp8, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp8, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp8, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 9){
            fprintf(kernel3_fp9, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp9, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp9, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp9, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 10){
            fprintf(kernel3_fp10, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp10, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp10, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp10, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 11){
            fprintf(kernel3_fp11, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp11, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp11, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp11, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 12){
            fprintf(kernel3_fp12, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp12, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp12, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp12, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 13){
            fprintf(kernel3_fp13, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp13, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp13, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp13, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 14){
            fprintf(kernel3_fp14, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp14, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp14, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp14, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 15){
            fprintf(kernel3_fp15, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp15, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp15, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp15, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 16){
            fprintf(kernel3_fp16, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp16, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp16, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp16, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 17){
            fprintf(kernel3_fp17, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp17, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp17, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp17, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 18){
            fprintf(kernel3_fp18, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp18, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp18, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp18, "%.16f\n", pagerank_array2[i]);
        }
        if(i == 19){
            fprintf(kernel3_fp19, "pagerank1\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp19, "%.16f\n", pagerank_array[i]);
            fprintf(kernel3_fp19, "pagerank2\n");
            for (int i = 0; i < num_nodes; i++)
                fprintf(kernel3_fp19, "%.16f\n", pagerank_array2[i]);
        }
        // Launch pagerank kernel 2
        pagerank2 <<<grid, threads>>>(pagerank1_d, pagerank2_d, num_nodes);
        sprintf(file_name, "kernel3_output%d.txt", i);
        err = hipMemcpy(pagerank_array , pagerank1_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        err = hipMemcpy(pagerank_array2, pagerank2_d, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
        FILE* kernel3_out = fopen(file_name, "w");
        fprintf(kernel3_out, "pagerank1\n");
        for (int j = 0; j < num_nodes; j++)
            fprintf(kernel3_out, "%.16f\n", pagerank_array[j]);
        fprintf(kernel3_out, "pagerank2\n");
        for (int j = 0; j < num_nodes; j++)
            fprintf(kernel3_out, "%.16f\n", pagerank_array2[j]);
        fclose(kernel3_out);
    }
    hipDeviceSynchronize();
    hipProfilerStop();
    double timer4 = gettime();
    fclose(kernel2_fp0);
    fclose(kernel2_fp1);
    fclose(kernel2_fp2);
    fclose(kernel2_fp3);
    fclose(kernel2_fp4);
    fclose(kernel2_fp5);
    fclose(kernel2_fp6);
    fclose(kernel2_fp7);
    fclose(kernel2_fp8);
    fclose(kernel2_fp9);
    fclose(kernel2_fp10);
    fclose(kernel2_fp11);
    fclose(kernel2_fp12);
    fclose(kernel2_fp13);
    fclose(kernel2_fp14);
    fclose(kernel2_fp15);
    fclose(kernel2_fp16);
    fclose(kernel2_fp17);
    fclose(kernel2_fp18);
    fclose(kernel2_fp19);

    fclose(kernel3_fp0);
    fclose(kernel3_fp1);
    fclose(kernel3_fp2);
    fclose(kernel3_fp3);
    fclose(kernel3_fp4);
    fclose(kernel3_fp5);
    fclose(kernel3_fp6);
    fclose(kernel3_fp7);
    fclose(kernel3_fp8);
    fclose(kernel3_fp9);
    fclose(kernel3_fp10);
    fclose(kernel3_fp11);
    fclose(kernel3_fp12);
    fclose(kernel3_fp13);
    fclose(kernel3_fp14);
    fclose(kernel3_fp15);
    fclose(kernel3_fp16);
    fclose(kernel3_fp17);
    fclose(kernel3_fp18);
    fclose(kernel3_fp19);
    // Copy the rank buffer back
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy() failed (%s)\n", hipGetErrorString(err));
        return -1;
    }

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    double timer2 = gettime();

    // Report timing characteristics
    printf("kernel time = %lf ms\n", (timer4 - timer3) * 1000);
    printf("kernel + memcpy time = %lf ms\n", (timer2 - timer1) * 1000);

#if 1
    // Print rank array
    print_vectorf(pagerank_array, num_nodes);
#endif

    // Free the host-side arrays
    free(pagerank_array);
    free(pagerank_array2);
    csr->freeArrays();
    free(csr);

    // Free the device buffers
    hipFree(row_d);
    hipFree(col_d);
    hipFree(data_d);

    hipFree(pagerank1_d);
    hipFree(pagerank2_d);

    return 0;

}

void print_vectorf(float *vector, int num)
{
    FILE * fp = fopen("result.out", "w");
    if (!fp) {
        printf("ERROR: unable to open result.txt\n");
    }

    for (int i = 0; i < num; i++) {
        fprintf(fp, "%f\n", vector[i]);
    }

    fclose(fp);
}

